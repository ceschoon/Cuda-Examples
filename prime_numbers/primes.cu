
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__
void areDivisible(int n, int Nb, int np, int *knownprimes, bool *ans)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x; // between [0 and Nb[
	int ni = n+i;  // number to be tested
	
	if (i<Nb)
	{
		ans[i] = false;
		for (int j=0; j<np; j++) 
		{
			int p = knownprimes[j];
			if (ni%p==0) ans[i] = true;
			
			// check we do not test further than sqrt(ni)
			if (p*p>ni) break;
		}
	}
}


bool isDivisible(int n, int np, int *primes)
{
	for (int i=0; i<np; i++) 
	{
		int p = primes[i];
		if (n%p==0) return true;
		
		// check we do not test further than sqrt(n)
		if (p*p>n) break;
	}
	
	return false;
}


int main(void)
{
	int Np = 1<<21;   // number of primes to compute
	int Nb = 1<<17;    // batch size of numbers tested at once on gpu
	
	int *primes, *d_primes;
	bool *ans, *d_ans;
	primes = (int*) malloc(Np*sizeof(int));
	ans = (bool*) malloc(Nb*sizeof(bool));
	hipMalloc(&d_primes, Np*sizeof(int));
	hipMalloc(&d_ans, Nb*sizeof(bool));
	
	// init
	int n=2;
	int np=0;
	
	// serial search for primes between 0 and Nb
	while(n<Nb)
	{
		// test if n is prime by checking division by previous primes
		if (!isDivisible(n,np,primes))
		{
			primes[np] = n;
			np ++; 
		}
		
		// next number
		n ++;
	}
	
	// parallel search for remaining primes
	while(np<Np)
	{
		// run divisibility tests on the batch from [n to n+Nb[
		hipMemcpy(d_primes, primes, np*sizeof(int), hipMemcpyHostToDevice);
		areDivisible<<<(Nb+255)/256, 256>>>(n, Nb, np, d_primes, d_ans);
		hipMemcpy(ans, d_ans, Nb*sizeof(bool), hipMemcpyDeviceToHost);
		
		// analyse results
		for (int i=0; i<Nb; i++) if (!ans[i] && np<Np)
		{
			primes[np] = n+i;
			np ++;
		}
		
		// increment
		n += Nb;
	}
	
	for (int i=0; i<Np; i++) printf("%d\n", primes[i]);
	
	hipFree(d_primes);
	hipFree(d_ans);
	free(primes);
	free(ans);
}
